#include "hip/hip_runtime.h"
#include "apra_cudamallochost_allocator.h"


	char* apra_cudamallochost_allocator::malloc(const size_type bytes)
	{
		void *ptr;
		auto errorCode = hipHostMalloc(&ptr, bytes);

		if (errorCode != hipSuccess)
		{
			// failed to allocate memory
			return NULL;
		}

		return reinterpret_cast<char *>(ptr);
	}
	
	void apra_cudamallochost_allocator::free(char *const block)
	{
		auto errorCode = hipHostFree(block);
		if (errorCode != hipSuccess)
		{
			// log error
		}
	}