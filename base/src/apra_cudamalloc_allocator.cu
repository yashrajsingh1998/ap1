#include "hip/hip_runtime.h"
#include "apra_cudamalloc_allocator.h"



	char* apra_cudamalloc_allocator::malloc(const size_type bytes)
	{
		void *ptr;
		auto errorCode = hipMalloc(&ptr, bytes);

		if (errorCode != hipSuccess)
		{
			// failed to allocate memory
			return NULL;
		}

		return reinterpret_cast<char *>(ptr);
	}
	
	void apra_cudamalloc_allocator::free(char *const block)
	{
		auto errorCode = hipFree(block);
		if (errorCode != hipSuccess)
		{
			// log error
		}
	}